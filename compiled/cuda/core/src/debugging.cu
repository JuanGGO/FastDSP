#include "hip/hip_runtime.h"
#include "../include/debugging.cuh"
#include "../include/cuda_helper.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>


namespace fdsp
{
#pragma region kernels
    __global__ void
    KernelPrintDevicePtr(const float *d_ptr, int N)
    {
        for(int i = 0; i < N; i++)
        {
            printf("%f ", d_ptr[i]);
            if((i+1) % 20 == 0)
                printf("\n");
        }
    }

    __global__ void
    KernelPrintDevicePtr(const double *d_ptr, int N)
    {
        for(int i = 0; i < N; i++)
        {
            printf("%f ", d_ptr[i]);
            if((i+1) % 20 == 0)
                printf("\n");
        }
    }

    __global__ void
    KernelPrintDevicePtr(const hipComplex *d_ptr, int N)
    {
        for(int i = 0; i < N; i++)
        {
            printf("%f + i%f  ", d_ptr[i].x, d_ptr[i].y);
            if ((i + 1) % 10 == 0)
                printf("\n");
        }
    }

#pragma endregion

#pragma region Host API

    void PrintDevicePtr(const float* d_ptr, int N)
    {
        KernelPrintDevicePtr<<<1, 1>>>(d_ptr, N);
        CHECK(hipDeviceSynchronize())
        CHECK(hipGetLastError());
    }

    void PrintDevicePtr(const double *d_ptr, int N)
    {
        KernelPrintDevicePtr<<<1, 1>>>(d_ptr, N);
        CHECK(hipDeviceSynchronize())
        CHECK(hipGetLastError());
    }

    void PrintDevicePtr(const hipComplex* d_ptr, int N)
    {
        KernelPrintDevicePtr<<<1, 1>>>(d_ptr, N);
        CHECK(hipDeviceSynchronize())
        CHECK(hipGetLastError());
    }

#pragma endregion

}