#include "../include/initialization.cuh"
#include <hip/hip_runtime_api.h>
#include "../include/cuda_helper.h"

int fdsp::GetDeviceCount()
{
    int nDevices;
    CHECK(hipGetDeviceCount(&nDevices));
    return nDevices;
}
