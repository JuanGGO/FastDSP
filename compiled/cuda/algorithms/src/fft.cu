#include "../include/fft_related.cuh"
#include "../core/include/cuda_helper.h"
#include <hipfft/hipfft.h>


namespace fdsp
{

    GPUArray<hipComplex> FourierTransform(const GPUArray<float>& array, int axis)
    {
        int rank = array.ndims;
        int *n = (int*)malloc(sizeof(int)*rank);
        std::vector<size_t> temp = array.GetDimensionSizes();
        for(int i = 0; i < rank; i++)
            n[i] = (int)temp[i];
        int size = array.GetSize();
        GPUArray<hipComplex> out(array.GetDimensionSizes());

        hipfftHandle plan;
        CHECK_CUFFT(hipfftCreate(&plan));
        switch (axis)
        {
            case -1:
            {
                CHECK_CUFFT(hipfftPlanMany(&plan, rank, (int*)n,
                                NULL, 1, size,
                                NULL, 1, size,
                                HIPFFT_R2C, 1));
                break;
            }
            case 0:
                CHECK_CUFFT(hipfftPlanMany(&plan, rank, (int*)n,
                                                (int*)n, n[0], 1, (int*)n, n[0],
                                                1, HIPFFT_R2C, n[rank-1]));
                break;
            default:
                break;
        }

        CHECK_CUFFT(hipfftExecR2C(plan, const_cast<float*>(array.GetPointerToArrayConst()), out.GetPointerToArray()));
        CHECK(hipDeviceSynchronize());

        CHECK_CUFFT(hipfftDestroy(plan));
        free(n);

        return out;
    }
}