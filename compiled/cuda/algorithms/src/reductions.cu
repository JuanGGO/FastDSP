#include "../include/basic_operations.cuh"
#include <stdexcept>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


template<typename T>
float fdsp::GetMean(const T *d_pArray, const std::vector<size_t>& dimSizes, int axis)
{
    if(axis > (int)dimSizes.size() - 1)
        throw std::runtime_error("Array of dimension " + std::to_string(dimSizes.size()) + " has no axis " + std::to_string(axis));

    float tmp;
    int size = 1;
    for(auto& sz: dimSizes)
        size *= sz;

    if(axis == -1)
    {
        tmp = thrust::reduce(thrust::device, d_pArray, d_pArray + size, 0.0, thrust::plus<float>());
        tmp /= size;
    }
    float out = static_cast<float>(tmp);

    return out;
}

template float fdsp::GetMean<unsigned char>(const unsigned char *d_pArray, const std::vector<size_t> &dimSizes, int axis);
template float fdsp::GetMean<int>(const int *d_pArray, const std::vector<size_t> &dimSizes, int axis);
template float fdsp::GetMean<float>(const float *d_pArray, const std::vector<size_t> &dimSizes, int axis);
template float fdsp::GetMean<double>(const double *d_pArray, const std::vector<size_t> &dimSizes, int axis);


template<typename T>
struct addComplex
{
    __host__ __device__
    T operator()(const T& c1, const T& c2)
    {
        T out;
        out.x = c1.x + c2.x;
        out.y = c1.y + c2.y;
        return out;
    }
};

template<typename T>
hipComplex fdsp::GetMeanComplex(const T* d_pArray, const std::vector<size_t>& dimSizes, int axis)
{
    T tmp;
    int size = 1;
    for(auto& sz: dimSizes)
        size *= sz;

    T start = {0, 0};
    tmp = thrust::reduce(thrust::device, d_pArray, d_pArray + size, start, addComplex<T>());
    tmp.x /= size;
    tmp.y /= size;

    hipComplex out;
    out.x = static_cast<float>(tmp.x);
    out.y = static_cast<float>(tmp.y);

    return out;
}

template hipComplex fdsp::GetMeanComplex<hipComplex>(const hipComplex* d_pArray, const std::vector<size_t>& dimSizes, int axis);
template hipComplex fdsp::GetMeanComplex<hipDoubleComplex>(const hipDoubleComplex* d_pArray, const std::vector<size_t>& dimSizes, int axis);
