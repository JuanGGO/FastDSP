#include "../include/data_structures.cuh"
#include "../core/include/debugging.cuh"
#include <hip/hip_complex.h>

#pragma region GPUArray

template<typename T>
fdsp::GPUArray<T>::GPUArray(const T* ptr, const std::vector<size_t>& dimSizes):
        m_dimSizes(dimSizes), ndims(dimSizes.size())
{
    m_size = 1;
    for(auto& size: m_dimSizes)
        m_size *= size;

    m_dArray.resize(m_size);
    CHECK(hipMemcpy(GetPointerToArray(), ptr, sizeof(T)*m_size, hipMemcpyHostToDevice));
}

template<typename T>
fdsp::GPUArray<T>::GPUArray(const std::vector<size_t> &dimSizes) :
    m_dimSizes(dimSizes), ndims(dimSizes.size())
{
    m_size = 1;
    for(auto& size: m_dimSizes)
        m_size *= size;

    m_dArray.resize(m_size);
    CHECK(hipMemset(GetPointerToArray(), 0, sizeof(T)*m_size));
}

template<typename T>
fdsp::GPUArray<T>::GPUArray(const GPUArray<T>& array)
{
    m_dimSizes = array.m_dimSizes;
    ndims = array.ndims;
    m_size = array.m_size;
    m_dArray.resize(m_size);
    CHECK(hipMemcpy(GetPointerToArray(), thrust::raw_pointer_cast(&array.m_dArray[0]), sizeof(T)*m_size, hipMemcpyDeviceToDevice));
}

template<typename T>
void fdsp::GPUArray<T>::Get(T *h_ptr) const
{
    thrust::copy(m_dArray.begin(), m_dArray.end(), h_ptr);
}

template<typename T>
T fdsp::GPUArray<T>::GetElement(size_t index) const
{
    return m_dArray[index];
}

template<typename T>
std::vector<size_t> fdsp::GPUArray<T>::GetDimensionSizes() const
{
    return m_dimSizes;
}

template<typename T>
const T* fdsp::GPUArray<T>::GetPointerToArrayConst()const
{
    return thrust::raw_pointer_cast(&(m_dArray[0]));
}

template<typename T>
T* fdsp::GPUArray<T>::GetPointerToArray()
{
    return thrust::raw_pointer_cast(&(m_dArray[0]));
}

template<typename T>
size_t fdsp::GPUArray<T>::GetSize() const
{
    return m_size;
}

template<typename T>
thrust::device_vector<T> fdsp::GPUArray<T>::GetDeviceVector() const
{
    return m_dArray;
}

template class fdsp::GPUArray<unsigned char>;
template class fdsp::GPUArray<int>;
template class fdsp::GPUArray<float>;
template class fdsp::GPUArray<double>;
template class fdsp::GPUArray<hipComplex>;
template class fdsp::GPUArray<hipDoubleComplex>;
#pragma endregion







