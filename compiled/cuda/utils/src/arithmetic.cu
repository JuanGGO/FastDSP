#include "hip/hip_runtime.h"
#include "../include/math.cuh"
#include<thrust/transform.h>
#include "../core/include/debugging.cuh"

#pragma region addition

struct Plus
{
    __device__
    float operator()(const float& x, const float& y)
    {
        return x + y;
    }
    __device__
    float operator()(const double& x, const double& y)
    {
        return x + y;
    }
    __device__
    hipComplex operator()(const hipComplex &x, const hipComplex &y)
    {
        hipComplex out;
        out.x = x.x + y.x;
        out.y = x.y + y.y;
        return out;
    }
    __device__
    hipDoubleComplex operator()(const hipDoubleComplex &x, const hipDoubleComplex &y)
    {
        hipDoubleComplex out;
        out.x = x.x + y.x;
        out.y = x.y + y.y;
        return out;
    }
};

template<typename T>
void fdsp::AddDevicePointers(const T *d_arr1, const T *d_arr2, int N, T *out)
{
        thrust::transform(thrust::device, d_arr1, d_arr1 + N, d_arr2, out, Plus());
}

template void fdsp::AddDevicePointers<float>(const float *d_arr1, const float *d_arr2, int N, float *out);
template void fdsp::AddDevicePointers<double>(const double *d_arr1, const double *d_arr2, int N, double *out);
template void fdsp::AddDevicePointers<hipComplex>(const hipComplex *d_arr1, const hipComplex *d_arr2, int N, hipComplex *out);
template void fdsp::AddDevicePointers<hipDoubleComplex>(const hipDoubleComplex *d_arr1, const hipDoubleComplex *d_arr2, int N, hipDoubleComplex *out);

template<typename T>
fdsp::GPUArray<T> fdsp::AddGPUArrays(const GPUArray<T>& arr1, const GPUArray<T>& arr2)
{
    GPUArray<T> out(arr1.GetDimensionSizes());
    fdsp::AddDevicePointers<T>(arr1.GetPointerToArrayConst(), arr2.GetPointerToArrayConst(),
                                arr1.GetSize(), out.GetPointerToArray());
    return out;
}

template fdsp::GPUArray<float> fdsp::AddGPUArrays<float>(const GPUArray<float>&, const GPUArray<float>&);
template fdsp::GPUArray<double> fdsp::AddGPUArrays<double>(const GPUArray<double>&, const GPUArray<double>&);
template fdsp::GPUArray<hipComplex> fdsp::AddGPUArrays<hipComplex>(const GPUArray<hipComplex>&, const GPUArray<hipComplex>&);
template fdsp::GPUArray<hipDoubleComplex> fdsp::AddGPUArrays<hipDoubleComplex>(const GPUArray<hipDoubleComplex>&, const GPUArray<hipDoubleComplex>&);

template<typename T>
void fdsp::AddGPUArrays(const GPUArray<T>& arr1, const GPUArray<T>& arr2, GPUArray<T>& out)
{
    fdsp::AddDevicePointers<T>(arr1.GetPointerToArrayConst(), arr2.GetPointerToArrayConst(),
                               arr1.GetSize(), out.GetPointerToArray());

}

template void fdsp::AddGPUArrays<float>(const GPUArray<float>&, const GPUArray<float>&, GPUArray<float>&);
template void fdsp::AddGPUArrays<double>(const GPUArray<double>&, const GPUArray<double>&, GPUArray<double>&);
template void fdsp::AddGPUArrays<hipComplex>(const GPUArray<hipComplex>&, const GPUArray<hipComplex>&, GPUArray<hipComplex>&);
template void fdsp::AddGPUArrays<hipDoubleComplex>(const GPUArray<hipDoubleComplex>&, const GPUArray<hipDoubleComplex>&, GPUArray<hipDoubleComplex>&);

#pragma endregion

#pragma region subtraction

struct Subtraction
{
    __device__
    float operator()(const float& x, const float& y)
    {
        return x - y;
    }
    __device__
    float operator()(const double& x, const double& y)
    {
        return x - y;
    }
    __device__
    hipComplex operator()(const hipComplex &x, const hipComplex &y)
    {
        hipComplex out;
        out.x = x.x - y.x;
        out.y = x.y - y.y;
        return out;
    }
    __device__
    hipDoubleComplex operator()(const hipDoubleComplex &x, const hipDoubleComplex &y)
    {
        hipDoubleComplex out;
        out.x = x.x - y.x;
        out.y = x.y - y.y;
        return out;
    }
};

template<typename T>
void fdsp::SubtractDevicePointers(const T *d_arr1, const T *d_arr2, int N, T *out)
{
    thrust::transform(thrust::device, d_arr1, d_arr1 + N, d_arr2, out, Subtraction());
}

template void fdsp::SubtractDevicePointers<float>(const float *d_arr1, const float *d_arr2, int N, float *out);
template void fdsp::SubtractDevicePointers<double>(const double *d_arr1, const double *d_arr2, int N, double *out);
template void fdsp::SubtractDevicePointers<hipComplex>(const hipComplex *d_arr1, const hipComplex *d_arr2, int N, hipComplex *out);
template void fdsp::SubtractDevicePointers<hipDoubleComplex>(const hipDoubleComplex *d_arr1, const hipDoubleComplex *d_arr2, int N, hipDoubleComplex *out);

template<typename T>
fdsp::GPUArray<T> fdsp::SubtractGPUArrays(const GPUArray<T>& arr1, const GPUArray<T>& arr2)
{
    GPUArray<T> out(arr1.GetDimensionSizes());
    fdsp::SubtractDevicePointers<T>(arr1.GetPointerToArrayConst(), arr2.GetPointerToArrayConst(),
                               arr1.GetSize(), out.GetPointerToArray());
    return out;
}

template fdsp::GPUArray<float> fdsp::SubtractGPUArrays<float>(const GPUArray<float>&, const GPUArray<float>&);
template fdsp::GPUArray<double> fdsp::SubtractGPUArrays<double>(const GPUArray<double>&, const GPUArray<double>&);
template fdsp::GPUArray<hipComplex> fdsp::SubtractGPUArrays<hipComplex>(const GPUArray<hipComplex>&, const GPUArray<hipComplex>&);
template fdsp::GPUArray<hipDoubleComplex> fdsp::SubtractGPUArrays<hipDoubleComplex>(const GPUArray<hipDoubleComplex>&, const GPUArray<hipDoubleComplex>&);

template<typename T>
void fdsp::SubtractGPUArrays(const GPUArray<T>& arr1, const GPUArray<T>& arr2, GPUArray<T>& out)
{
    fdsp::SubtractDevicePointers<T>(arr1.GetPointerToArrayConst(), arr2.GetPointerToArrayConst(),
                               arr1.GetSize(), out.GetPointerToArray());

}

template void fdsp::SubtractGPUArrays<float>(const GPUArray<float>&, const GPUArray<float>&, GPUArray<float>&);
template void fdsp::SubtractGPUArrays<double>(const GPUArray<double>&, const GPUArray<double>&, GPUArray<double>&);
template void fdsp::SubtractGPUArrays<hipComplex>(const GPUArray<hipComplex>&, const GPUArray<hipComplex>&, GPUArray<hipComplex>&);
template void fdsp::SubtractGPUArrays<hipDoubleComplex>(const GPUArray<hipDoubleComplex>&, const GPUArray<hipDoubleComplex>&, GPUArray<hipDoubleComplex>&);

#pragma endregion subtraction

#pragma region multiplication

struct Multiplication
{
    __device__
    float operator()(const float& x, const float& y)
    {
        return x*y;
    }
    __device__
    float operator()(const double& x, const double& y)
    {
        return x*y;
    }
    __device__
    hipComplex operator()(const hipComplex &x, const hipComplex &y)
    {
        hipComplex out;
        out.x = x.x*y.x - x.y*y.y;
        out.y = x.x*y.y + x.y*y.x;
        return out;
    }
    __device__
    hipDoubleComplex operator()(const hipDoubleComplex &x, const hipDoubleComplex &y)
    {
        hipDoubleComplex out;
        out.x = x.x*y.x - x.y*y.y;
        out.y = x.x*y.y + x.y*y.x;
        return out;
    }
};

template<typename T>
void fdsp::MultiplyDevicePointers(const T *d_arr1, const T *d_arr2, int N, T *out)
{
    thrust::transform(thrust::device, d_arr1, d_arr1 + N, d_arr2, out, Multiplication());
}

template void fdsp::MultiplyDevicePointers<float>(const float *d_arr1, const float *d_arr2, int N, float *out);
template void fdsp::MultiplyDevicePointers<double>(const double *d_arr1, const double *d_arr2, int N, double *out);
template void fdsp::MultiplyDevicePointers<hipComplex>(const hipComplex *d_arr1, const hipComplex *d_arr2, int N, hipComplex *out);
template void fdsp::MultiplyDevicePointers<hipDoubleComplex>(const hipDoubleComplex *d_arr1, const hipDoubleComplex *d_arr2, int N, hipDoubleComplex *out);

template<typename T>
fdsp::GPUArray<T> fdsp::MultiplyGPUArrays(const GPUArray<T>& arr1, const GPUArray<T>& arr2)
{
    GPUArray<T> out(arr1.GetDimensionSizes());
    fdsp::MultiplyDevicePointers<T>(arr1.GetPointerToArrayConst(), arr2.GetPointerToArrayConst(),
                                    arr1.GetSize(), out.GetPointerToArray());
    return out;
}

template fdsp::GPUArray<float> fdsp::MultiplyGPUArrays<float>(const GPUArray<float>&, const GPUArray<float>&);
template fdsp::GPUArray<double> fdsp::MultiplyGPUArrays<double>(const GPUArray<double>&, const GPUArray<double>&);
template fdsp::GPUArray<hipComplex> fdsp::MultiplyGPUArrays<hipComplex>(const GPUArray<hipComplex>&, const GPUArray<hipComplex>&);
template fdsp::GPUArray<hipDoubleComplex> fdsp::MultiplyGPUArrays<hipDoubleComplex>(const GPUArray<hipDoubleComplex>&, const GPUArray<hipDoubleComplex>&);

template<typename T>
void fdsp::MultiplyGPUArrays(const GPUArray<T>& arr1, const GPUArray<T>& arr2, GPUArray<T>& out)
{
    fdsp::MultiplyDevicePointers<T>(arr1.GetPointerToArrayConst(), arr2.GetPointerToArrayConst(),
                                    arr1.GetSize(), out.GetPointerToArray());

}

template void fdsp::MultiplyGPUArrays<float>(const GPUArray<float>&, const GPUArray<float>&, GPUArray<float>&);
template void fdsp::MultiplyGPUArrays<double>(const GPUArray<double>&, const GPUArray<double>&, GPUArray<double>&);
template void fdsp::MultiplyGPUArrays<hipComplex>(const GPUArray<hipComplex>&, const GPUArray<hipComplex>&, GPUArray<hipComplex>&);
template void fdsp::MultiplyGPUArrays<hipDoubleComplex>(const GPUArray<hipDoubleComplex>&, const GPUArray<hipDoubleComplex>&, GPUArray<hipDoubleComplex>&);

#pragma endregion

#pragma region division

struct Division
{
    __device__
    float operator()(const float& x, const float& y)
    {
        return x/y;
    }
    __device__
    float operator()(const double& x, const double& y)
    {
        return x/y;
    }
    __device__
    hipComplex operator()(const hipComplex &x, const hipComplex &y)
    {
        hipComplex out;
        float norm = y.x*y.x + y.y*y.y;
        out.x = (x.x*y.x + x.y*y.y)/norm;
        out.y = (x.y*y.x - x.x*y.y)/norm;
        return out;
    }
    __device__
    hipDoubleComplex operator()(const hipDoubleComplex &x, const hipDoubleComplex &y)
    {
        hipDoubleComplex out;
        double norm = y.x*y.x + y.y*y.y;
        out.x = (x.x*y.x + x.y*y.y)/norm;
        out.y = (x.y*y.x - x.x*y.y)/norm;
        return out;
    }
};

template<typename T>
void fdsp::DivideDevicePointers(const T *d_arr1, const T *d_arr2, int N, T *out)
{
    thrust::transform(thrust::device, d_arr1, d_arr1 + N, d_arr2, out, Division());
}

template void fdsp::DivideDevicePointers<float>(const float *d_arr1, const float *d_arr2, int N, float *out);
template void fdsp::DivideDevicePointers<double>(const double *d_arr1, const double *d_arr2, int N, double *out);
template void fdsp::DivideDevicePointers<hipComplex>(const hipComplex *d_arr1, const hipComplex *d_arr2, int N, hipComplex *out);
template void fdsp::DivideDevicePointers<hipDoubleComplex>(const hipDoubleComplex *d_arr1, const hipDoubleComplex *d_arr2, int N, hipDoubleComplex *out);

template<typename T>
fdsp::GPUArray<T> fdsp::DivideGPUArrays(const GPUArray<T>& arr1, const GPUArray<T>& arr2)
{
    GPUArray<T> out(arr1.GetDimensionSizes());
    fdsp::DivideDevicePointers<T>(arr1.GetPointerToArrayConst(), arr2.GetPointerToArrayConst(),
                                    arr1.GetSize(), out.GetPointerToArray());
    return out;
}

template fdsp::GPUArray<float> fdsp::DivideGPUArrays<float>(const GPUArray<float>&, const GPUArray<float>&);
template fdsp::GPUArray<double> fdsp::DivideGPUArrays<double>(const GPUArray<double>&, const GPUArray<double>&);
template fdsp::GPUArray<hipComplex> fdsp::DivideGPUArrays<hipComplex>(const GPUArray<hipComplex>&, const GPUArray<hipComplex>&);
template fdsp::GPUArray<hipDoubleComplex> fdsp::DivideGPUArrays<hipDoubleComplex>(const GPUArray<hipDoubleComplex>&, const GPUArray<hipDoubleComplex>&);

template<typename T>
void fdsp::DivideGPUArrays(const GPUArray<T>& arr1, const GPUArray<T>& arr2, GPUArray<T>& out)
{
    fdsp::DivideDevicePointers<T>(arr1.GetPointerToArrayConst(), arr2.GetPointerToArrayConst(),
                                    arr1.GetSize(), out.GetPointerToArray());

}

template void fdsp::DivideGPUArrays<float>(const GPUArray<float>&, const GPUArray<float>&, GPUArray<float>&);
template void fdsp::DivideGPUArrays<double>(const GPUArray<double>&, const GPUArray<double>&, GPUArray<double>&);
template void fdsp::DivideGPUArrays<hipComplex>(const GPUArray<hipComplex>&, const GPUArray<hipComplex>&, GPUArray<hipComplex>&);
template void fdsp::DivideGPUArrays<hipDoubleComplex>(const GPUArray<hipDoubleComplex>&, const GPUArray<hipDoubleComplex>&, GPUArray<hipDoubleComplex>&);

#pragma endregion
